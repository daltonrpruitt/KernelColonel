// driver.cpp
// simple driver file for kernel testing

#define DEBUG
// local files
#include <driver.h>
#include <device_props.h>
#include <copy.cu>
#include <simple_indirection.cu>
#include <overlap_index_access_with_data.cu>
#include <computation.cu>
#include <output.h>
#include <utils.h>

#include <iostream>
#include <string>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>



using vt = double;
using std::cout;
using std::endl;
using std::string;
using std::to_string;

#define N (32*32*32 * 32 * 8)

int main() {
    timespec mainStart, mainEnd;
    clock_gettime(CLOCK_MONOTONIC, &mainStart);
    int total_runs = 0;

    cout << "Processing " << N  << " elements" << endl;
    typedef ArrayCopyContext<vt, int> copy_kernel_t;
    typedef MicrobenchmarkDriver<copy_kernel_t> copy_driver_t;

    typedef MicrobenchmarkDriver<SimpleIndirectionKernel<vt, int, false>> indirection_driver_direct_t;
    typedef MicrobenchmarkDriver<SimpleIndirectionKernel<vt, int, true>> indirection_driver_indirect_t;
    
    typedef MicrobenchmarkDriver<OverlappedIdxDataAccessKernel<vt, int, 1>> overlapped_access_driver_1_t;
    typedef MicrobenchmarkDriver<OverlappedIdxDataAccessKernel<vt, int, 2>> overlapped_access_driver_2_t;
    typedef MicrobenchmarkDriver<OverlappedIdxDataAccessKernel<vt, int, 4>> overlapped_access_driver_4_t;
    typedef MicrobenchmarkDriver<OverlappedIdxDataAccessKernel<vt, int, 8>> overlapped_access_driver_8_t;

    device_context dev_ctx;
    if(!dev_ctx.init()) return -1;

    std::vector<int> bs_vec;
    // Only one of the next two lines 
    // for (int bs = 32; bs <= 1024; bs *= 2) { bs_vec.push_back(bs);}
    bs_vec.push_back(128);

    Output output_dir;
    if(output_dir.empty()) {
        cerr << "Not continuing!" << endl;
        return -1;
    }
    string filename = "new_file.csv";
    cout << output_dir+"indirect_kernel_output.csv" << endl;
    cout << output_dir+filename << endl;

    copy_driver_t copy_driver(N, bs_vec, output_dir+"copy_kernel_output.csv", &dev_ctx, true);
    if (!copy_driver.check_then_run_kernels()) {return -1;} 
    total_runs += copy_driver.get_total_runs();

    
    indirection_driver_direct_t direct_driver(N, bs_vec,output_dir+"direct_kernel_output.csv", &dev_ctx, true);
    if (!direct_driver.check_then_run_kernels()) {return -1;} 
    indirection_driver_indirect_t indirect_driver(N, bs_vec, output_dir+"indirect_kernel_output.csv", &dev_ctx, true);
    if (!indirect_driver.check_then_run_kernels()) {return -1;} 
    total_runs += direct_driver.get_total_runs() + indirect_driver.get_total_runs();


    overlapped_access_driver_1_t overlapped_1_driver(N, bs_vec, output_dir+"overlapped_1_kernel_output.csv", &dev_ctx, true);
    if (!overlapped_1_driver.check_then_run_kernels()) {return -1;} 
    overlapped_access_driver_2_t overlapped_2_driver(N, bs_vec, output_dir+"overlapped_2_kernel_output.csv", &dev_ctx, true);
    if (!overlapped_2_driver.check_then_run_kernels()) {return -1;} 
    overlapped_access_driver_4_t overlapped_4_driver(N, bs_vec, output_dir+"overlapped_4_kernel_output.csv", &dev_ctx, true);
    if (!overlapped_4_driver.check_then_run_kernels()) {return -1;} 
    overlapped_access_driver_8_t overlapped_8_driver(N, bs_vec, output_dir+"overlapped_8_kernel_output.csv", &dev_ctx, true);
    if (!overlapped_8_driver.check_then_run_kernels()) {return -1;} 
    total_runs += overlapped_1_driver.get_total_runs() + overlapped_2_driver.get_total_runs() + 
                            overlapped_4_driver.get_total_runs() + overlapped_8_driver.get_total_runs();


    MicrobenchmarkDriver<ComputationalIntensityContext<vt, int, 1>> comp_intens_1_driver(N, bs_vec, output_dir+"computational_intensity_1_kernel_output.csv", &dev_ctx, true);
    if (!comp_intens_1_driver.check_then_run_kernels()) {return -1;} 
    MicrobenchmarkDriver<ComputationalIntensityContext<vt, int, 2>> comp_intens_2_driver(N, bs_vec, output_dir+"computational_intensity_2_kernel_output.csv", &dev_ctx, true);
    if (!comp_intens_2_driver.check_then_run_kernels()) {return -1;} 
    MicrobenchmarkDriver<ComputationalIntensityContext<vt, int, 4>> comp_intens_4_driver(N, bs_vec, output_dir+"computational_intensity_4_kernel_output.csv", &dev_ctx, true);
    if (!comp_intens_4_driver.check_then_run_kernels()) {return -1;} 
    MicrobenchmarkDriver<ComputationalIntensityContext<vt, int, 8>> comp_intens_8_driver(N, bs_vec, output_dir+"computational_intensity_8_kernel_output.csv", &dev_ctx, true);
    if (!comp_intens_8_driver.check_then_run_kernels()) {return -1;} 
    MicrobenchmarkDriver<ComputationalIntensityContext<vt, int, 16>> comp_intens_16_driver(N, bs_vec, output_dir+"computational_intensity_16_kernel_output.csv", &dev_ctx, true);
    if (!comp_intens_16_driver.check_then_run_kernels()) {return -1;} 
    MicrobenchmarkDriver<ComputationalIntensityContext<vt, int, 32>> comp_intens_32_driver(N, bs_vec, output_dir+"computational_intensity_32_kernel_output.csv", &dev_ctx, true);
    if (!comp_intens_32_driver.check_then_run_kernels()) {return -1;} 
    MicrobenchmarkDriver<ComputationalIntensityContext<vt, int, 64>> comp_intens_64_driver(N, bs_vec, output_dir+"computational_intensity_64_kernel_output.csv", &dev_ctx, true);
    if (!comp_intens_64_driver.check_then_run_kernels()) {return -1;} 

    total_runs += comp_intens_1_driver.get_total_runs() + comp_intens_2_driver.get_total_runs() + 
                            comp_intens_4_driver.get_total_runs() + comp_intens_8_driver.get_total_runs() + 
                            comp_intens_16_driver.get_total_runs() + comp_intens_32_driver.get_total_runs() + 
                            comp_intens_64_driver.get_total_runs();


    clock_gettime(CLOCK_MONOTONIC, &mainEnd);
    double main_time = elapsed_time_ms(mainStart, mainEnd);
    
    cout << "#########  Finished  #########" << endl << endl;
    cout << "Total runs performed        = " << total_runs << endl;
    cout << "Total time taken (m:ss)     = " <<(int)main_time / 1000 / 60 << ":" << (int)main_time / 1000 % 60 << endl;

    return 0;
}