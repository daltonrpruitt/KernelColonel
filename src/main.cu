#include "hip/hip_runtime.h"
// driver.cpp
// simple driver file for kernel testing

#define DEBUG
// local files
#include <driver.h>
#include <device_props.h>
#include <kernels/general/copy.cu>
#include <kernels/general/simple_indirection.cu>
#include <kernels/general/overlap_index_access_with_data.cu>
#include <kernels/general/computation.cu>
#include <kernels/burst_mode/interleaved_copy.cu>
#include <kernels/uncoalesced_cached_access/uncoalesced_reuse.cu>

#include <output.h>
#include <utils.h>

#include <iostream>
#include <string>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>



using vt = double;
using std::cout;
using std::endl;
using std::string;
using std::to_string;

// #define N (32*32*32 * 32 * 8)

int main() {
    timespec mainStart, mainEnd;
    clock_gettime(CLOCK_MONOTONIC, &mainStart);
    int total_runs = 0;

    typedef ArrayCopyContext<vt, int> copy_kernel_t;
    typedef MicrobenchmarkDriver<copy_kernel_t> copy_driver_t;

    typedef MicrobenchmarkDriver<SimpleIndirectionKernel<vt, int, false>> indirection_driver_direct_t;
    typedef MicrobenchmarkDriver<SimpleIndirectionKernel<vt, int, true>> indirection_driver_indirect_t;
    
    typedef MicrobenchmarkDriver<OverlappedIdxDataAccessKernel<vt, int, 1>> overlapped_access_driver_1_t;
    typedef MicrobenchmarkDriver<OverlappedIdxDataAccessKernel<vt, int, 2>> overlapped_access_driver_2_t;
    typedef MicrobenchmarkDriver<OverlappedIdxDataAccessKernel<vt, int, 4>> overlapped_access_driver_4_t;
    typedef MicrobenchmarkDriver<OverlappedIdxDataAccessKernel<vt, int, 8>> overlapped_access_driver_8_t;

    device_context dev_ctx;
    if(!dev_ctx.init()) return -1;
    unsigned long long min_array_size = dev_ctx.props_.l2CacheSize / sizeof(vt) * 10;
    unsigned long long N = pow(2, ceil(log2(min_array_size)));

    cout << "Processing " << N  << " elements" << endl;


    std::vector<int> bs_vec;
    // Only one of the next two lines 
    for (int bs = 256; bs <= 1024; bs *= 2) { bs_vec.push_back(bs);}
    // bs_vec.push_back(128);

    Output output_dir;
    if(output_dir.empty()) {
        cerr << "Not continuing!" << endl;
        return -1;
    }

    // copy_driver_t copy_driver(N, bs_vec, output_dir+"copy_kernel_output.csv", &dev_ctx, true);
    // if (!copy_driver.check_then_run_kernels()) {return -1;} 
    // total_runs += copy_driver.get_total_runs();

    
    // indirection_driver_direct_t direct_driver(N, bs_vec,output_dir+"direct_kernel_output.csv", &dev_ctx, true);
    // if (!direct_driver.check_then_run_kernels()) {return -1;} 
    // indirection_driver_indirect_t indirect_driver(N, bs_vec, output_dir+"indirect_kernel_output.csv", &dev_ctx, true);
    // if (!indirect_driver.check_then_run_kernels()) {return -1;} 
    // total_runs += direct_driver.get_total_runs() + indirect_driver.get_total_runs();


    // overlapped_access_driver_1_t overlapped_1_driver(N, bs_vec, output_dir+"overlapped_1_kernel_output.csv", &dev_ctx, true);
    // if (!overlapped_1_driver.check_then_run_kernels()) {return -1;} 
    // overlapped_access_driver_2_t overlapped_2_driver(N, bs_vec, output_dir+"overlapped_2_kernel_output.csv", &dev_ctx, true);
    // if (!overlapped_2_driver.check_then_run_kernels()) {return -1;} 
    // overlapped_access_driver_4_t overlapped_4_driver(N, bs_vec, output_dir+"overlapped_4_kernel_output.csv", &dev_ctx, true);
    // if (!overlapped_4_driver.check_then_run_kernels()) {return -1;} 
    // overlapped_access_driver_8_t overlapped_8_driver(N, bs_vec, output_dir+"overlapped_8_kernel_output.csv", &dev_ctx, true);
    // if (!overlapped_8_driver.check_then_run_kernels()) {return -1;} 
    // total_runs += overlapped_1_driver.get_total_runs() + overlapped_2_driver.get_total_runs() + 
    //                         overlapped_4_driver.get_total_runs() + overlapped_8_driver.get_total_runs();


    // MicrobenchmarkDriver<ComputationalIntensityContext<vt, int, 1>> comp_intens_1_driver(N, bs_vec, output_dir+"computational_intensity_1_kernel_output.csv", &dev_ctx, true);
    // if (!comp_intens_1_driver.check_then_run_kernels()) {return -1;} 
    // MicrobenchmarkDriver<ComputationalIntensityContext<vt, int, 2>> comp_intens_2_driver(N, bs_vec, output_dir+"computational_intensity_2_kernel_output.csv", &dev_ctx, true);
    // if (!comp_intens_2_driver.check_then_run_kernels()) {return -1;} 
    // MicrobenchmarkDriver<ComputationalIntensityContext<vt, int, 4>> comp_intens_4_driver(N, bs_vec, output_dir+"computational_intensity_4_kernel_output.csv", &dev_ctx, true);
    // if (!comp_intens_4_driver.check_then_run_kernels()) {return -1;} 
    // MicrobenchmarkDriver<ComputationalIntensityContext<vt, int, 8>> comp_intens_8_driver(N, bs_vec, output_dir+"computational_intensity_8_kernel_output.csv", &dev_ctx, true);
    // if (!comp_intens_8_driver.check_then_run_kernels()) {return -1;} 
    // MicrobenchmarkDriver<ComputationalIntensityContext<vt, int, 16>> comp_intens_16_driver(N, bs_vec, output_dir+"computational_intensity_16_kernel_output.csv", &dev_ctx, true);
    // if (!comp_intens_16_driver.check_then_run_kernels()) {return -1;} 
    // MicrobenchmarkDriver<ComputationalIntensityContext<vt, int, 32>> comp_intens_32_driver(N, bs_vec, output_dir+"computational_intensity_32_kernel_output.csv", &dev_ctx, true);
    // if (!comp_intens_32_driver.check_then_run_kernels()) {return -1;} 
    // MicrobenchmarkDriver<ComputationalIntensityContext<vt, int, 64>> comp_intens_64_driver(N, bs_vec, output_dir+"computational_intensity_64_kernel_output.csv", &dev_ctx, true);
    // if (!comp_intens_64_driver.check_then_run_kernels()) {return -1;} 

    // total_runs += comp_intens_1_driver.get_total_runs() + comp_intens_2_driver.get_total_runs() + 
    //                         comp_intens_4_driver.get_total_runs() + comp_intens_8_driver.get_total_runs() + 
    //                         comp_intens_16_driver.get_total_runs() + comp_intens_32_driver.get_total_runs() + 
    //                         comp_intens_64_driver.get_total_runs();
    

#define INTER_DRIVER(X, Y) interleaved_copy_ ## X  ## _ ## Y ## _driver
#define XSTRINGIFY( x ) STRINGIFY ( x )
#define STRINGIFY( x ) #x

#define INTERLEAVED(X, Y) MicrobenchmarkDriver<InterleavedCopyContext<vt, int, X, Y>> \
      INTER_DRIVER(X, Y)(N, bs_vec, output_dir+ XSTRINGIFY( INTER_DRIVER(X, Y) ) ".csv", &dev_ctx, true); \
    if (!INTER_DRIVER(X, Y).check_then_run_kernels()) {return -1;}  \
    total_runs += INTER_DRIVER(X, Y).get_total_runs();

   
    
    INTERLEAVED(1, 1)
    INTERLEAVED(2, 1)
    INTERLEAVED(4, 1)
    INTERLEAVED(8, 1)
    INTERLEAVED(16, 1)
    INTERLEAVED(32, 1)
    // INTERLEAVED(64, 1)

    // INTERLEAVED(1, 1)
    INTERLEAVED(1, 2)
    INTERLEAVED(1, 4)
    INTERLEAVED(1, 8)
    INTERLEAVED(1, 16)
    INTERLEAVED(1, 32)
    // INTERLEAVED(1, 64)

    // INTERLEAVED(8, 1, 1)
    INTERLEAVED(8, 2)
    INTERLEAVED(8, 4)
    INTERLEAVED(8, 8)
    INTERLEAVED(8, 16)
    INTERLEAVED(8, 32)

    // INTERLEAVED(32, 1)
    INTERLEAVED(32, 2)
    INTERLEAVED(32, 4)
    INTERLEAVED(32, 8)
    INTERLEAVED(32, 16)


#define UNCOAL_REUSE_DRIVER(B1, B2) uncoalesced_reuse_ ## B1  ## _ ## B2 ## _driver

#define UNCOAL_REUSE(B1, B2) MicrobenchmarkDriver<UncoalescedReuseContext<vt, int, B1, B2>> \
      UNCOAL_REUSE_DRIVER(B1, B2)(N, bs_vec, output_dir+ XSTRINGIFY( UNCOAL_REUSE_DRIVER(B1, B2) ) ".csv", &dev_ctx, true); \
    if (!UNCOAL_REUSE_DRIVER(B1, B2).check_then_run_kernels()) {return -1;}  \
    total_runs += UNCOAL_REUSE_DRIVER(B1, B2).get_total_runs();
    
    // UNCOAL_REUSE(false, false)
    // UNCOAL_REUSE(true, false)
    // UNCOAL_REUSE(false, true)
    // UNCOAL_REUSE(true, true)





    clock_gettime(CLOCK_MONOTONIC, &mainEnd);
    double main_time = elapsed_time_ms(mainStart, mainEnd);
    
    cout << "#########  Finished  #########" << endl << endl;
    cout << "Total runs performed        = " << total_runs << endl;
    cout << "Total time taken (h:mm:ss)     = " << std::setfill('0') << std::setw(2) 
                                                << (int)main_time / 1000 / 60 / 60 << ":" 
                                                << std::setfill('0') << std::setw(2)
                                                << (int)main_time / 1000 / 60 << ":" 
                                                << std::setfill('0') << std::setw(2)
                                                << (int)main_time / 1000 % 60 << endl;

    return 0;
}
