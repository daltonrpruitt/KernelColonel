#include "hip/hip_runtime.h"
#pragma once
/**
 * @file spmv_la_1.cu
 * @author Dalton Winans-Pruitt (daltonrpruitt@gmail.com)
 * @brief Derived from TemplateKernelContext
 * @version 0.1
 * @date 2022-04-07
 * 
 * This SpMV kernel is to test using a version of latency amortization
 * that is fairly simple: loading in chunks of the vector into cache. 
 *
 */

#include <iostream>
#include <iomanip>
#include <vector>
#include <string>
#include <typeinfo>   // operator typeid
#include <filesystem>

#include <hip/hip_runtime.h>
#include <local_cuda_utils.h>
#include <crs_mat.h>
#include <kernels/spmv/spmv_base.cu>

#define DEBUG

using std::string;
using std::cout;
using std::endl;
using std::vector;
namespace fs = std::filesystem;

#ifndef WARP_SIZE
#define WARP_SIZE (32)
#endif

template <typename vt=double>
__forceinline__ __host__ __device__
void force_global_load(vt* arr, uint offset, uint m) {
    if(offset >= m) return; 
    vt tmp_vec;
    // https://www.cplusplus.com/reference/typeinfo/type_info/operator==/
    if constexpr(std::is_same<vt,double>()) {
        asm volatile("ld.global.f64 %0, [%1];"
                    : "=d"(tmp_vec) : "l"((vt *)(arr + offset)));
    } else if constexpr(std::is_same<vt,float>()) {
        asm volatile("ld.global.f32 %0, [%1];"
                    : "=f"(tmp_vec) : "l"((vt *)(arr + offset)));
    } else {
        static_assert(std::is_same<vt,double>()); // Know will fail at this point, but needed to get around ill-formed argument https://stackoverflow.com/questions/38304847/constexpr-if-and-static-assert
    }
    return;
}

#define MAX_THREADS_PER_BLOCK 64
#define MIN_BLOCKS_PER_MP     2

template <typename it=int, typename vt=double, int ILP = 1>
__launch_bounds__(MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP)
// __forceinline__ __host__ __device__ 
__global__ 
void spmv_kernel_latency_amortization_1(vt* product, CRSMat_gpu matrix, vt* vec) {
    uint g_t_id = blockIdx.x * blockDim.x + threadIdx.x;
    uint warp_id = g_t_id / WARP_SIZE;
    if(warp_id >= matrix.m) return;
    // uint stride = 2 * 32 / sizeof(vt);
    uint lane = threadIdx.x % WARP_SIZE; 
    // assume vector is preloaded into cache

#if __CUDA_ARCH__ >= 700
    uint stride = 2 * WARP_SIZE / sizeof(vt); // 2 sectors
#else
    uint stride = 1 * WARP_SIZE / sizeof(vt); // 1 sector
#endif

    // uint row_id = warp_id;
    uint start = matrix.offsets[warp_id];
    uint stop =  matrix.offsets[warp_id + 1];
    uint vals_processed = stop - start;

    uint chunk_parts = 2;
    uint chunk_size = WARP_SIZE * chunk_parts;
    int num_chunks = (vals_processed + chunk_size) / chunk_size;

    vt t_sum = 0;
    for(int chunk=0; chunk < num_chunks; chunk++) {
        uint local_start = start + chunk * chunk_size;

        uint local_start_col_idx = matrix.indices[local_start];
        uint local_stop_col_idx = max(matrix.indices[min(local_start + chunk_size, stop)-1], local_start_col_idx + 1);
        uint cur_preload_start_idx = local_start_col_idx;
        while(cur_preload_start_idx < local_stop_col_idx) {
            force_global_load<vt>(vec, cur_preload_start_idx + lane*stride, local_stop_col_idx);
            cur_preload_start_idx += WARP_SIZE * stride;
        }
    
    
        for(uint part=0; part < chunk_parts; part++) {
            uint immediate_idx = local_start + part*WARP_SIZE + lane;
            if(immediate_idx >= stop) break;
            vt val = matrix.values[immediate_idx];
            it col = matrix.indices[immediate_idx];
            t_sum += val * vec[col];
        }
    }
    // if (lane == 0) { product[warp_id] = local_stop_col_idx;} return;
    
    // Final parallel reduce
    unsigned m = 0xffffffff;
    for (int offset = 16; offset > 0; offset /= 2) {
        t_sum += __shfl_down_sync(m, t_sum, offset);
    }
    if (lane == 0) {
        product[warp_id] = t_sum;  // Single thread writing single value...
    }
    return;
}

template <typename it=int, typename vt=double>
struct SpmvKernelLAv1 : SpmvKernel<it, vt> {
   public:
    typedef SpmvKernel<it, vt> super;


    SpmvKernelLAv1(int bs, device_context* d_ctx, int shd_mem_alloc = 0, int matrix_file_id=0) 
    : super(bs, d_ctx, shd_mem_alloc, matrix_file_id) {
        this->name = "SpmvKernelLAv1";
    }
    ~SpmvKernelLAv1() {}

    void output_config_info() override {
        cout << "SpMV Latency Amortization V1 with : "
                << " Bsz=" << this->Bsz 
                << " Blocks used ="<< this->Gsz
                << " occupancy=" << this->get_occupancy() << endl;

    }

    float local_execute() override {  
        if(this->dev_ctx->props_.major >= 7) {
            hipFuncAttributes attr;
            hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((void *)) spmv_kernel_latency_amortization_1<int, double>);
            int shmem = this->dev_ctx->props_.sharedMemPerMultiprocessor-1024-attr.sharedSizeBytes;
            hipFuncSetAttribute((void *) spmv_kernel_latency_amortization_1<int, double>, hipFuncAttributeMaxDynamicSharedMemorySize, shmem);
            hipFuncSetAttribute((void *) spmv_kernel_latency_amortization_1<int, double>, hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxShared);
            cudaPrintLastError();
        }
        hipEvent_t start, stop;
        hipEventCreate(&start); hipEventCreate(&stop);

        
        hipEventRecord(start);
        // dense_vector_cache_preload<<<preload_blocks, Bsz, shared_memory_usage>>>(gpu_vector, gpu_matrix.m);
        // hipDeviceSynchronize();
        // cudaPrintLastError();
        spmv_kernel_latency_amortization_1<it,vt><<<this->Gsz, this->Bsz, this->shared_memory_usage>>>(this->gpu_results, this->gpu_matrix, this->gpu_vector);
        hipEventRecord(stop);

        hipEventSynchronize(stop);
        cudaPrintLastError();

        float time = 0;
        hipEventElapsedTime(&time, start, stop);
        hipEventDestroy(start); hipEventDestroy(stop);

        return time; 
    }

    // No change
    void local_compute_register_usage(bool& pass) override {
        // Kernel Registers
        struct hipFuncAttributes funcAttrib;
        cudaErrChk(hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(*spmv_kernel_latency_amortization_1<it), vt>), "getting function attributes (for # registers)", pass);
        if (!pass) {
            this->okay = false;
            return;
        }
        this->register_usage = funcAttrib.numRegs;
    }

    string get_local_extra_config_parameters() override { 
        return "";
    
    string get_local_extra_config_values() { 
        return "";
    } 

};
