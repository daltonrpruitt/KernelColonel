#include "hip/hip_runtime.h"
#pragma once
/**
 * @file spmv_la_1.cu
 * @author Dalton Winans-Pruitt (daltonrpruitt@gmail.com)
 * @brief Derived from TemplateKernelContext
 * @version 0.1
 * @date 2022-04-07
 * 
 * This SpMV kernel is to test using a version of latency amortization
 * that is fairly simple: loading in chunks of the vector into cache. 
 *
 */

#include <iostream>
#include <iomanip>
#include <vector>
#include <string>
#include <typeinfo>   // operator typeid
#include <filesystem>

#include <hip/hip_runtime.h>
#include <local_cuda_utils.h>
#include <local_basic_utils.h>
#include <crs_mat.h>
#include <kernels/spmv/spmv_base.cu>

// #define DEBUG

using std::string;
using std::cout;
using std::endl;
using std::vector;
namespace fs = std::filesystem;

#ifndef WARP_SIZE
#define WARP_SIZE (32)
#endif

#define MAX_THREADS_PER_BLOCK 64
#define MIN_BLOCKS_PER_MP     2

template <typename it=int, typename vt=double, bool preload=false, bool include_preload_arith=false, int chunk_parts=1>
__launch_bounds__(MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP)
// __forceinline__ __host__ __device__ 
__global__ 
void spmv_kernel_latency_amortization_1(vt* product, CRSMat_gpu<it,vt> matrix, vt* vec) {
    unsigned int g_t_id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int warp_id = g_t_id / WARP_SIZE;
    if(warp_id >= matrix.m) return;
    // unsigned int stride = 2 * 32 / sizeof(vt);
    unsigned int lane = threadIdx.x % WARP_SIZE; 
    // assume vector is preloaded into cache

#if __CUDA_ARCH__ >= 700
    unsigned int stride = 2 * 32 / sizeof(vt); // 2 sectors
#else
    unsigned int stride = 1 * 32 / sizeof(vt); // 1 sector
#endif

    // unsigned int row_id = warp_id;
    unsigned int start = matrix.offsets[warp_id];
    unsigned int stop =  matrix.offsets[warp_id + 1];
    unsigned int vals_processed = stop - start;

    // unsigned int chunk_parts = 2;
    unsigned int chunk_size = WARP_SIZE * chunk_parts;
    int num_chunks = (vals_processed + chunk_size) / chunk_size;

    vt t_sum = 0;
    for(int chunk=0; chunk < num_chunks; chunk++) {
        unsigned int local_start = start + chunk * chunk_size;

        unsigned int local_start_col_idx=0, local_stop_col_idx=0, cur_preload_start_idx=local_start_col_idx;

        if constexpr(include_preload_arith || preload) {
            local_start_col_idx = matrix.indices[local_start];
            local_stop_col_idx = max(matrix.indices[min(local_start + chunk_size, stop)-1], local_start_col_idx + 1);
            cur_preload_start_idx = local_start_col_idx;
        }
        if constexpr(include_preload_arith || preload) {
            while(cur_preload_start_idx < local_stop_col_idx) {
                if constexpr(preload){
                    force_global_load<vt>(vec, cur_preload_start_idx + lane*stride, local_stop_col_idx);
                }
                cur_preload_start_idx += WARP_SIZE * stride;
            }
        }
    
    
        for(unsigned int part=0; part < chunk_parts; part++) {
            unsigned int immediate_idx = local_start + part*WARP_SIZE + lane;
            if(immediate_idx >= stop) break;
            vt val = matrix.values[immediate_idx];
            it col = matrix.indices[immediate_idx];
            t_sum += val * vec[col];
        }
    }
    // if (lane == 0) { product[warp_id] = local_stop_col_idx;} return;
    
    // Final parallel reduce
    unsigned m = 0xffffffff;
    for (int offset = 16; offset > 0; offset /= 2) {
        t_sum += __shfl_down_sync(m, t_sum, offset);
    }
    if (lane == 0) {
        product[warp_id] = t_sum;  // Single thread writing single value...
    }
    return;
}

template <typename it=int, typename vt=double, bool preload=false, bool include_preload_arith=false, int chunk_parts=1>
struct SpmvKernelLAv1 : SpmvKernel<it, vt> {
   public:
    typedef SpmvKernel<it, vt> super;


  SpmvKernelLAv1(int bs, device_context* d_ctx, string mtx_filename, int shd_mem_alloc = 0) 
    : super(bs, d_ctx, mtx_filename, shd_mem_alloc) {
        this->name = "SpmvKernelLAv1";
        static_assert(!( preload && !include_preload_arith)); // if preload, must include arith (just to reduce test space complexity)
    }
    ~SpmvKernelLAv1() {}

    void output_config_info() override {
        cout << "SpMV Latency Amortization V1 with : "
                << "\n\t Bsz=" << this->Bsz 
                << "\n\t Blocks used ="<< this->Gsz
                << "\n\t matrix file="<< fs::path(this->matrix_filename).filename()
                << "\n\t occupancy=" << this->get_occupancy()
                << "\n\t preload=" << bool_to_string(preload)
                << "\n\t include_preload_arithmetic=" << bool_to_string(include_preload_arith)
                << "\n\t chunk_parts=" << chunk_parts << endl;
    }

    float local_execute() override {  
        if(this->dev_ctx->props_.major >= 7) {
            hipFuncAttributes attr;
            hipFuncGetAttributes(&attr, 
                reinterpret_cast<const void*>((void *)) spmv_kernel_latency_amortization_1<it, vt, preload, include_preload_arith, chunk_parts>);
            int shmem = this->dev_ctx->props_.sharedMemPerMultiprocessor-1024-attr.sharedSizeBytes;
            hipFuncSetAttribute(
                (void *) spmv_kernel_latency_amortization_1<it, vt, preload, include_preload_arith, chunk_parts>, 
                hipFuncAttributeMaxDynamicSharedMemorySize, shmem);
            hipFuncSetAttribute(
                (void *) spmv_kernel_latency_amortization_1<it, vt, preload, include_preload_arith, chunk_parts>, 
                hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxShared);
            cudaPrintLastError();
        }
        hipEvent_t start, stop;
        hipEventCreate(&start); hipEventCreate(&stop);

        
        hipEventRecord(start);
        // dense_vector_cache_preload<<<preload_blocks, Bsz, shared_memory_usage>>>(gpu_vector, gpu_matrix.m);
        // hipDeviceSynchronize();
        // cudaPrintLastError();
        spmv_kernel_latency_amortization_1<it,vt,preload,include_preload_arith,chunk_parts>
            <<<this->Gsz, this->Bsz, this->shared_memory_usage>>>(this->gpu_results, this->gpu_matrix, this->gpu_vector);
        hipEventRecord(stop);

        hipEventSynchronize(stop);
        cudaPrintLastError();

        float time = 0;
        hipEventElapsedTime(&time, start, stop);
        hipEventDestroy(start); hipEventDestroy(stop);

        return time; 
    }

    // No change
    void local_compute_register_usage(bool& pass) override {
        // Kernel Registers
        struct hipFuncAttributes funcAttrib;
        cudaErrChk(hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(spmv_kernel_latency_amortization_1<it),vt,preload,include_preload_arith,chunk_parts>), "getting function attributes (for # registers)", pass);
        if (!pass) {
            this->okay = false;
            return;
        }
        this->register_usage = funcAttrib.numRegs;
    }

    string get_local_extra_config_parameters() override { 
        return "preload,include_preload_arith,chunk_parts";
    }
    
    string get_local_extra_config_values() { 
        stringstream out;
        out << bool_to_string(preload) << "," << bool_to_string(include_preload_arith) << "," << chunk_parts;
        return out.str();
    } 

};
