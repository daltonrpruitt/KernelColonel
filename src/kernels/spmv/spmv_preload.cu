#include "hip/hip_runtime.h"
#pragma once
/**
 * @file spmv_preload.cu
 * @author Dalton Winans-Pruitt (daltonrpruitt@gmail.com)
 * @brief Derived from TemplateKernelContext
 * @version 0.1
 * @date 2022-02-08
 * 
 * This application is for performing sparse matrix-vector 
 * multiplication (SpMV) operations.
 *
 */

#include <iostream>
#include <iomanip>
#include <vector>
#include <string>

#include <hip/hip_runtime.h>
#include <local_cuda_utils.h>
#include <crs_mat.h>
#include <kernel_context.cu>

#define DEBUG

using std::string;
using std::cout;
using std::endl;
using std::vector;

template <typename it=int, typename vt=double, int ILP = 1>
// __forceinline__ __host__ __device__
__global__ 
void dense_vector_cache_preload(vt* vector, int m) {
    uint g_t_id = blockIdx.x * blockDim.x + threadIdx.x;
    // uint warp_id = g_t_id / warpSize;
    uint stride = 2 * 32 / sizeof(vt);

    // assume m % stride == 0
    if (g_t_id < m / stride) {
        vt tmp_vec;  // = vector[g_t_id*stride];
        asm volatile("ld.global.f64 %0, [%1];"
                     : "=d"(tmp_vec) : "l"((double *)(vector + g_t_id * stride)));
        //if(tmp_vec == -0.0001) return;  // not needed if asm volatile (?)
    }
    return;
}

template <typename it=int, typename vt=double, int ILP = 1>
// __forceinline__ __host__ __device__ 
__global__ 
void spmv_kernel(vt* product, CRSMat_gpu matrix, vt* vector) { //}, int max_nz_row) {
    uint g_t_id = blockIdx.x * blockDim.x + threadIdx.x;
    uint warp_id = g_t_id / warpSize;
    if(warp_id >= matrix.m) return;
    // uint stride = 2 * 32 / sizeof(vt);
    uint lane = threadIdx.x % warpSize; 
    // assume vector is preloaded into cache

    // uint row_id = warp_id;
    uint start = matrix.offsets[warp_id];
    uint stop =  matrix.offsets[warp_id + 1];
    uint vals_processed = stop - start;

    // if (lane == 0) {
    //     product[warp_id] =  (vals_processed / warpSize) + 1;
    // }
    // return;

    vt t_sum = 0;
    // assume n >> blockDim.x (5000 >> 128 or 256-ish)
    for (int i = 0; i < (vals_processed / warpSize) + 1; ++i) {
        if (lane + i * warpSize >= vals_processed) break;
        vt val = matrix.values[ start + i * warpSize + lane];
        it col = matrix.indices[start + i * warpSize + lane];
        t_sum += val * vector[col];
    }
    unsigned m = 0xffffffff;
    for (int offset = 16; offset > 0; offset /= 2) {
        t_sum += __shfl_down_sync(m, t_sum, offset);
    }
    if (lane == 0) {
        product[warp_id] = t_sum;  // Single thread writing single value...
    }
    return;
}

std::vector<string> matrix_filenames= {
    "../../matrices/bcsstk13.mtx"
};


template <typename it=int, typename vt=double>
class SpmvKernel {
   public:
    // typedef KernelCPUContext<vt, it> super;
    string name = "spmv";
    bool initialized = false, okay = true;
    int Bsz, Gsz;


    size_t shared_memory_usage=0;
    int register_usage=-1;
    int max_blocks_simultaneous_per_sm=-1;
    device_context* dev_ctx;


    uint matrix_id;
    CRSMat host_matrix;
    CRSMat_gpu gpu_matrix;
    uint nnz;
    vector<double> host_vector;
    double *gpu_vector;
    vector<double> host_results;
    double *gpu_results;
    
    /*  
  //int N = super::N;
  //int Gsz = super::Gsz;
  //int Bsz = super::Bsz;
  
        vector<vt> & n = super::host_data[0];
        vector<vt> & out = super::host_data[1];
        vt* & d_in = super::device_data_ptrs[0];
        vt* & d_out = super::device_data_ptrs[1];
  */
    int data_reads_per_element = -1;   // Actual number
    int index_reads_per_element = -1;  // Actual number
    int writes_per_element = -1;       // Actual number

    // struct gpu_ctx {
    //     // Params for kernel; set in set_dev_ptrs()
    //     CRSMat gpu_mat;
    //     vt* gpu_vector;
    //     vt* gpu_results;
    //     vt* gpu_out;

    //     __device__ void operator()(uint idx, vt* product, CRSMat matrix, vt* vector, int max_nz_row) {
    //         extern __shared__ int dummy[];
    //         kernel<vt, it>(idx, product,  matrix, vector, max_nz_row);
    //     }
    // } ctx;

    SpmvKernel(int n, int bs, device_context* d_ctx, int shd_mem_alloc = 0, int run_times = 25, int matrix_file_id=0) 
    : Bsz(bs), Gsz( (n+bs-1)/bs ), dev_ctx(d_ctx), shared_memory_usage(shd_mem_alloc), host_matrix(matrix_filenames[matrix_file_id]) {
        //  : super(2, 1, 2, n, bs, dev_ctx, shd_mem_alloc) {
        //this->name = "SpMV";
        // this->total_data_reads = N * data_reads_per_element;
        // this->total_index_reads = N * index_reads_per_element;
        // this->total_writes = N * writes_per_element;

        if(!check() ) {return;}


    }
    ~SpmvKernel() { uninit(); }

    void init(bool& pass)  {
        // Init Matrix here (host arrays/data)
        // host_matrix = CRSMat();
        if(host_matrix.nnz < 0) {
            pass = false; 
            return;
        }
        #ifdef DEBUG
        host_matrix.dump();
        #endif

        gpu_matrix.nnz = host_matrix.nnz;
        gpu_matrix.m   = host_matrix.m;
        gpu_matrix.n   = host_matrix.n;

        host_vector.reserve(host_matrix.m);
        for(int i=0; i < host_matrix.m; ++i) { host_vector[i] = (vt)i; }
        host_results.reserve(host_matrix.m);
        for(int i=0; i < host_matrix.m; ++i) { host_results[i] = (vt)0; }

        /* !!!!!!!!!!!!!!!!!!!!!!!*/
        // Allocate gpu arrays/copy host to gpu
        cudaErrChk(hipMalloc((void **)&gpu_matrix.values,gpu_matrix.nnz * sizeof(double)),"gpu_matrix.values mem allocation", pass);
        if(pass){
            cudaErrChk(hipMalloc((void **)&gpu_matrix.indices,gpu_matrix.nnz * sizeof(int)),"gpu_matrix.indices mem allocation", pass);
        }	
        if(pass){
            cudaErrChk(hipMalloc((void **)&gpu_matrix.offsets,(gpu_matrix.m+1) * sizeof(int)),"gpu_matrix.offsets mem allocation", pass);
        }
        if(pass){
            cudaErrChk(hipMalloc((void **)&gpu_vector,(gpu_matrix.m) * sizeof(double)),"gpu_vector mem allocation", pass);
        }
        if(pass){
            cudaErrChk(hipMalloc((void **)&gpu_results,(gpu_matrix.m) * sizeof(double)),"gpu_vector mem allocation", pass);
        }
        
        // cudaMemCpy 
        if(pass) {
            cudaErrChk(
                hipMemcpy(gpu_matrix.values, host_matrix.values,gpu_matrix.nnz * sizeof(double), hipMemcpyHostToDevice),
                "copy host_matrix.values to gpu_matrix.values", pass
                );
        }
        
        if(pass){
            cudaErrChk(
                hipMemcpy(gpu_matrix.indices, host_matrix.indices, gpu_matrix.nnz * sizeof(int), hipMemcpyHostToDevice),
                "copy host_matrix.indices to gpu_matrix.indices", pass
                );
        }	
        if(pass){
            cudaErrChk(
                hipMemcpy(gpu_matrix.offsets, host_matrix.offsets, (gpu_matrix.m+1) * sizeof(int), hipMemcpyHostToDevice),
                "copy host_matrix.offsets to gpu_matrix.offsets", pass
                );
        }
        if(pass){
            cudaErrChk(
                hipMemcpy(gpu_vector,host_vector.data(),gpu_matrix.m * sizeof(double), hipMemcpyHostToDevice),
                "copy host_vector to gpu_vector", pass
                );
        }
        if(pass){
            cudaErrChk(
                hipMemset(gpu_results, 0, gpu_matrix.m *sizeof(double)),
                "initializing gpu_results to 0", pass
                );
        }


        if (!pass) {
            cerr << "Could not initialize " << name << "!" << endl;
            return;
        }

        initialized = true;
        return;
    }

    void uninit() {
        // 8. Uninitialize data on device and host
        if (!initialized) {
            return;
        }

        // CudaFree gpu memory

        // Update this section!
        delete host_matrix.values; host_matrix.values = nullptr;
        delete host_matrix.indices; host_matrix.indices = nullptr;
        delete host_matrix.offsets; host_matrix.offsets = nullptr;
        vector<double>().swap(host_vector); 
        vector<double>().swap(host_results); 

        hipFree(gpu_matrix.values); gpu_matrix.values = nullptr;
        hipFree(gpu_matrix.indices); gpu_matrix.indices = nullptr;
        hipFree(gpu_matrix.offsets); gpu_matrix.offsets = nullptr;
        hipFree(gpu_vector); gpu_vector = nullptr;
        hipFree(gpu_results); gpu_results = nullptr;

        initialized = false;
    }

    bool local_check_result() {
        // Perform matrix multiply here
        bool debug = false;
#ifdef DEBUG
        debug = true;
#endif
        vector<double> cpu_results;
        for(int i=0; i < host_matrix.m; ++i) { cpu_results.push_back(0); }
        
        int cur_row_start = 0;
        for(int i=0; i < host_matrix.m; i++) {
            if(debug && i < 10) cout << "Row " << i << ": "; 
            double result = 0;
            int start = host_matrix.offsets[i];
            int end   = host_matrix.offsets[i+1];
            int row_nz = end - start;
            for(int j=0; j < row_nz; j++) {
                int cur_pos = cur_row_start + j; 
                int col = host_matrix.indices[cur_pos];
                double val = host_matrix.values[cur_pos];
                double vec_val = host_vector[col];
                
                if(debug && i < 10 && j < 32) cout <<  std::setprecision(2) << val << "*" << std::setprecision(2) << vec_val << ", "; 
                result += val * vec_val; 

            }
            if(debug && i < 10) cout <<  "...  = " << std::setprecision(2) << result << endl; 
            cpu_results[i] = result;
            
            cur_row_start += row_nz;
        }

        for(int i=0; i < host_matrix.m; i++) {
            if(abs(cpu_results[i] - host_results[i])/cpu_results[i] > 1e-3 ) {
                cout << "Results are incorrect at " << i << ": host=" << std::setprecision(3) << cpu_results[i] 
                    << " device="  << std::setprecision(3) << host_results[i] << endl;
                int output_num = 10;
                int print_start = max(0, i-output_num / 2);
                int print_end = min(host_matrix.m, i+output_num / 2);
                for(int j=print_start; j < print_end; ++j) {
                    cout << "\t" << j << ": " << std::setprecision(3) << cpu_results[j] 
                                    << " =?= "  << std::setprecision(3) << host_results[j] << endl;  
                }
                return false;
            }
        
        }

        // Check resulting vector with output of kernel
/*
        for (int i = 0; i < N; ++i) {
            if (in[i] != out[i]) {
                cout << "Validation Failed at " << i << ": in=" << in[i] << " out=" << out[i] << endl;
                return false;
            }
        }
        */
        return true;
    }
    
    bool check() {
        if(!initialized) {
            bool pass = true;
            init(pass);
            if(!pass) return false;
        }
        if(execute() < 0) return false;
        float time = local_check_result(); 
        uninit();
        return time;
    }

    float local_execute() {
        //  Need to update since will be using two separate kernels.

        //  return  local_execute_template<gpu_ctx>(N, Gsz, Bsz, this->shared_memory_usage, this->dev_ctx, ctx);
   
        if(dev_ctx->props_.major >= 7) {
            hipFuncAttributes attr;
            hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((void *)) dense_vector_cache_preload<int, double>);
            int shmem = dev_ctx->props_.sharedMemPerMultiprocessor-1024-attr.sharedSizeBytes;
            hipFuncSetAttribute((void *) dense_vector_cache_preload<int, double>, hipFuncAttributeMaxDynamicSharedMemorySize, shmem);
            hipFuncSetAttribute((void *) dense_vector_cache_preload<int, double>, hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxShared);
            cudaPrintLastError();

            hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((void *)) spmv_kernel<int, double>);
            shmem = dev_ctx->props_.sharedMemPerMultiprocessor-1024-attr.sharedSizeBytes;
            hipFuncSetAttribute((void *) spmv_kernel<int, double>, hipFuncAttributeMaxDynamicSharedMemorySize, shmem);
            hipFuncSetAttribute((void *) spmv_kernel<int, double>, hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxShared);
            cudaPrintLastError();


        }
        hipEvent_t start, stop;
        hipEventCreate(&start); hipEventCreate(&stop);

        int stride = 4;
        if(dev_ctx->props_.major >= 7) {
            stride = 8;
        }
        int preload_blocks = host_matrix.m / (Bsz / warp_size * stride) + 1; 
        int spmv_blocks = host_matrix.m / (Bsz / warp_size) + 1;
        
        hipEventRecord(start);
        dense_vector_cache_preload<<<preload_blocks, Bsz, shared_memory_usage>>>(gpu_vector, gpu_matrix.m);
        hipDeviceSynchronize();
        cudaPrintLastError();
        spmv_kernel<<<spmv_blocks, Bsz, shared_memory_usage>>>(gpu_results, gpu_matrix, gpu_vector);
        hipEventRecord(stop);

        hipEventSynchronize(stop);
        cudaPrintLastError();

        float time = 0;
        hipEventElapsedTime(&time, start, stop);
        hipEventDestroy(start); hipEventDestroy(stop);

        return time; 
    }

    void copy_result_to_host(bool &pass) {
        cout << "Before copy to host :" ; 
        for(int i=0; i < 64; ++i) { cout << " " << std::setprecision(2) << host_results[i]; if(i %32 == 31) { cout << endl << "\t";} }
        cout << endl;

        cudaErrChk(hipMemcpy(host_results.data(), gpu_results, host_matrix.m * sizeof(double), hipMemcpyDeviceToHost),"copying results from gpu to host", pass);

        cout << "After copy to host :" ; 
        for(int i=0; i < 64; ++i) { cout << " " << std::setprecision(2) << host_results[i]; if(i %32 == 31) { cout << endl << "\t";} }
        cout << endl;

    }   

    float execute() {
        if(!okay) return -1.0;

        float time = local_execute();
        bool pass = true;
        copy_result_to_host(pass);
        
        if(!pass) {uninit(); okay = false; time = -1.0;}
        return time;
    }

    float run() {
        if(!initialized) {
            bool pass = true;
            init(pass);
            if(!pass) return -1.0;
        }
        return execute();
    }

    // No change
    void local_compute_register_usage(bool& pass) {
        // Kernel Registers
        struct hipFuncAttributes funcAttrib;
        cudaErrChk(hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(*spmv_kernel<vt), it>), "getting function attributes (for # registers)", pass);
        if (!pass) {
            this->okay = false;
            return;
        }
        this->register_usage = funcAttrib.numRegs;
    }

    void compute_max_simultaneous_blocks(bool& pass) {
        local_compute_register_usage(pass);
        if (!pass) {
            okay = false;
            return;
        }
        int due_to_block_size = (int)floor(dev_ctx->props_.maxThreadsPerMultiProcessor / Bsz);
        int due_to_registers = (int)floor(dev_ctx->props_.regsPerMultiprocessor / (register_usage * Bsz));
        max_blocks_simultaneous_per_sm = std::min({due_to_block_size,
                                                   due_to_registers, dev_ctx->props_.maxBlocksPerMultiProcessor});
    }

    vector<int> shared_memory_allocations() {
        return vector<int>(0);
        /*
        vector<int> alloc_amounts;
        bool pass = true;
        if (max_blocks_simultaneous_per_sm < 0) compute_max_simultaneous_blocks(pass);
        if (!pass) {
            okay = false;
            alloc_amounts.push_back(-1);
            return alloc_amounts;
        }
        int max_shd_mem_per_block = dev_ctx->props_.sharedMemPerBlock;
        int max_shd_mem_per_proc = dev_ctx->props_.sharedMemPerMultiprocessor;
        if (dev_ctx->props_.major == 8) {
            max_shd_mem_per_block = max_shd_mem_per_proc;
        }
        int min_blocks_due_to_shd_mem = max_shd_mem_per_proc / max_shd_mem_per_block;

        for (int i = min_blocks_due_to_shd_mem; i < max_blocks_simultaneous_per_sm; i *= 2) {
            int sm_alloc = std::min((max_shd_mem_per_proc / i) / 256 * 256, max_shd_mem_per_block);
            if (dev_ctx->props_.major == 8) {
                sm_alloc -= 1024;
            }
            if (std::find(alloc_amounts.begin(), alloc_amounts.end(), sm_alloc) == alloc_amounts.end()) {
                alloc_amounts.push_back(sm_alloc);
            }
        }
        return alloc_amounts;
        */
    }

    float get_occupancy() {
        bool pass = true;
        if (max_blocks_simultaneous_per_sm < 0) compute_max_simultaneous_blocks(pass);
        if (!pass) {
            okay = false;
            return -1.0;
        }

        int max_blocks_shared_mem;
        if (shared_memory_usage == 0) {
            max_blocks_shared_mem = dev_ctx->props_.maxBlocksPerMultiProcessor;
        } else {
            max_blocks_shared_mem = dev_ctx->props_.sharedMemPerMultiprocessor / shared_memory_usage;
        }
        int max_blocks_simul = std::min(max_blocks_simultaneous_per_sm, max_blocks_shared_mem);
        int num_threads_simul = max_blocks_simul * Bsz;
        return float(num_threads_simul) / float(dev_ctx->props_.maxThreadsPerMultiProcessor);
    }

    int get_sharedmemory_from_occupancy(float occupancy) {
        bool pass = true;
        if (max_blocks_simultaneous_per_sm < 0) compute_max_simultaneous_blocks(pass);
        if (!pass) {
            okay = false;
            return -1;
        }

        int blocks = float(dev_ctx->props_.maxThreadsPerMultiProcessor / Bsz) * occupancy;
        if (blocks > max_blocks_simultaneous_per_sm) {
            cerr << "Try to get occupancy higher than architecture allows!" << endl;
            return -1;
        }

        int max_shd_mem_per_block = dev_ctx->props_.sharedMemPerBlock;
        int max_shd_mem_per_proc = dev_ctx->props_.sharedMemPerMultiprocessor;
        if (dev_ctx->props_.major == 8) {
            max_shd_mem_per_block = max_shd_mem_per_proc;
        }

        int shdmem = max_shd_mem_per_proc / blocks;

        if (shdmem > max_shd_mem_per_block) {
            cerr << "Cannot set shared memory high enough to match occupancy of " << occupancy << "!" << endl;
            shdmem = max_shd_mem_per_block;
        }
        if (dev_ctx->props_.major == 8) {
            shdmem -= 1024;
        }
        return shdmem;
    }

    void print_register_usage() {
        bool pass = true;
        if (register_usage < 0) {
            local_compute_register_usage(pass);
        }
        if (!pass) {
            cerr << "Cannot get register usage for " << name << "!" << endl;
        } else {
            cout << name << " register usage = " << register_usage << endl;
        }
    }

    unsigned long long get_total_bytes_processed() {
        //  return ( total_data_reads+ total_writes)*sizeof(vt) +  total_index_reads*sizeof(it);
    }
};
