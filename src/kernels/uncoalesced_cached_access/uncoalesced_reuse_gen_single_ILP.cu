#include "hip/hip_runtime.h"
#pragma once
/**
 * @file uncoalesced_reuse_gen_single_ILP.cu
 * @author Dalton Winans-Pruitt (daltonrpruitt@gmail.com)
 * @brief Based on UncoalescedReuseGeneralSingleElementContext; 
 *      testing for cache reducing impact of noncoalesced accesses  
 * @version 0.1
 * @date 2022-03-10
 * 
 * Adding ILP (instruction-level parallelism) to uncoalesced reuse kernel. 
 * 
 */

#include <iostream>
#include <vector>
#include <string>
#include <cassert>

#include <hip/hip_runtime.h>
#include <local_cuda_utils.h>
#include <kernel_context.cu>

using std::string;
using std::cout;
using std::endl;
using std::vector;


template<typename vt, typename it, bool preload_for_reuse, bool avoid_bank_conflicts, int shuffle_size, int ILP>
__forceinline__ __host__ __device__        
void uncoalesced_reuse_gen_single_ilp_kernel(uint idx, vt* gpu_in, vt* gpu_out, unsigned long long N){
void uncoalesced_reuse_general_single_kernel(uint idx, vt* gpu_in, vt* gpu_out, unsigned long long N){

    uint Sz = shuffle_size; 
    uint shuffle_b_idx = idx / Sz;
    uint shuffle_t_idx = idx % Sz;

    uint num_warps = shuffle_size / 32;
    
    // Preload data
    if constexpr(preload_for_reuse) {
        vt tmp = gpu_in[idx];
        if(tmp < 0) return; // all values should be > 0; this is just to ensure this write is not removed
    }

    int start_idx = shuffle_b_idx * Sz;

    unsigned long long access_idx;
    if constexpr(!avoid_bank_conflicts) {
        access_idx = ( shuffle_t_idx % num_warps) * 32 + shuffle_t_idx / num_warps + start_idx;
    } else {
        access_idx = ( (shuffle_t_idx % 32) * 32 + (shuffle_t_idx % 32 + shuffle_t_idx / num_warps ) % 32) % Sz + start_idx;
    }

    gpu_out[idx] = gpu_in[access_idx];
}


template<typename vt, typename it, bool preload_for_reuse, bool avoid_bank_conflicts, int shuffle_size, int ILP>
__global__        
void kernel_for_regs_reuse_gen_single_ilp(uint idx, vt* gpu_in, vt* gpu_out, unsigned long long N){
        extern __shared__ int dummy[];
        uncoalesced_reuse_gen_single_ilp_kernel<vt, it, preload_for_reuse, avoid_bank_conflicts, shuffle_size, ILP>(idx, gpu_in, gpu_out, N);
}

template<typename vt, typename it, bool preload_for_reuse, bool avoid_bank_conflicts, int shuffle_size, int ILP>
struct UncoalescedReuseGenSingleILPContext : public KernelCPUContext<vt, it> {
    public:
        typedef KernelCPUContext<vt, it> super;
        unsigned long long N = super::N;
        int Gsz = super::Gsz;
        int Bsz = super::Bsz;

        vector<vt> & in = super::host_data[0];
        vector<vt> & out = super::host_data[1];
        vt* & d_in = super::device_data_ptrs[0];
        vt* & d_out = super::device_data_ptrs[1];

        int data_reads_per_element = 1; // only one valualbe read?
        int index_reads_per_element = 0;
        int writes_per_element = 1;
        struct gpu_ctx {
            vt * gpu_in;
            vt * gpu_out;
            unsigned long long N;

            __device__        
            void operator() (uint idx){
                extern __shared__ int dummy[];
                uncoalesced_reuse_gen_single_ilp_kernel<vt, it, preload_for_reuse, avoid_bank_conflicts, shuffle_size, ILP>(idx, gpu_in, gpu_out, N);
            }
        } ctx ;

        UncoalescedReuseGenSingleILPContext(int n, int bs, device_context* dev_ctx, int shd_mem_alloc=0) 
            : super(1, 1, 0, n, bs, dev_ctx, shd_mem_alloc) {
            this->name = "UncoalescedReuseGenSingleILP"; 
            this->Gsz /= ILP;
            assert(this->Gsz > 0);
            // if constexpr(preload_for_reuse) {
            //     data_reads_per_element += 1;
            // }

            this->total_data_reads = N * data_reads_per_element;
            this->total_index_reads = N * index_reads_per_element;
            this->total_writes = N * writes_per_element;
        }
        ~UncoalescedReuseGenSingleILPContext(){}

        void init_inputs(bool& pass) override {
            for(int i=0; i<N; ++i){
                in.push_back(i);
                out.push_back(0);
            }
        }

        void init_indices(bool& pass) override {}

        void set_dev_ptrs() override {
            ctx.gpu_in = d_in;
            ctx.gpu_out = d_out;
            ctx.N = N;
        }

        void output_config_info() override {
            cout << this->name << " with : "
                 <<" preloading?=" << preload_for_reuse 
                 << " avoiding bank conflicts?=" << avoid_bank_conflicts << endl;
        }

        float local_execute() override {
            return local_execute_template<gpu_ctx>(N, Gsz, Bsz, this->shared_memory_usage, this->dev_ctx, ctx);
        }

        bool local_check_result() override {
            bool pass = true;
            int num_warps = shuffle_size / 32;
            unsigned long long global_tidx = 0;
            for(int i=0; i < this->N / shuffle_size; ++i) {
                int start_idx = i * shuffle_size;
                for (int j=0; j < shuffle_size; ++j){
                    global_tidx = start_idx + j;

                    uint shuffle_t_idx = global_tidx % shuffle_size;
                    vt val = 0;
                    if constexpr(!avoid_bank_conflicts) {
                        val = in[( shuffle_t_idx % num_warps) * 32 + shuffle_t_idx / num_warps + start_idx];
                    } else {
                        val = in[( (shuffle_t_idx % 32) * 32 + (shuffle_t_idx % 32 + shuffle_t_idx / num_warps ) % 32) % shuffle_size + start_idx];

                    }
                    if (out[global_tidx] != val) {
                        cout << "Validation Failed at " << global_tidx << ": in="<<in[global_tidx] << " out="<< out[global_tidx] << endl;
                        pass = false;
                        break;
                    }
                }
                if(!pass) break;
            }

            if(!pass) {
                cout << "Debug dump of in and out array: " << endl;
                cout << std::setw(10) << "IN" << "  |" << std::setw(10) << "OUT " << endl; 
                int output_size = 10;
                unsigned long long j = max((int)0, (int)(global_tidx - output_size/2));
                for(int k=0; k < output_size; ++k, ++j) { 
                    cout << std::setw(10) << in[j] <<"  |" <<std::setw(10)<<out[j] << endl; 
                }
            }
            return pass;
        }

        void local_compute_register_usage(bool& pass) override {   
            // Kernel Registers 
            struct hipFuncAttributes funcAttrib;
            cudaErrChk(hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(*kernel_for_regs_reuse_gen_single_ilp<vt),it,preload_for_reuse,avoid_bank_conflicts,shuffle_size,ILP>), "getting function attributes (for # registers)", pass);
            if(!pass) {
                this->okay = false; 
                return;
            }
            this->register_usage = funcAttrib.numRegs;
        }

};
