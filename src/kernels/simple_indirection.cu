#include "hip/hip_runtime.h"
#pragma once
/**
 * @file simple_indirection.cu
 * @author Dalton Winans-Pruitt (daltonrpruitt@gmail.com)
 * @brief 
 * @version 0.1
 * @date 2022-02-08
 * 
 * Based on copy.cu 
 * Most of what needs to be changed (or not) is listed.
 * Do not modify function signatures unless obvious (comment in signature).
 */

#include <vector>
#include <type_traits>

#include <hip/hip_runtime.h>
#include <local_cuda_utils.h>
#include <kernel_context.cu>

#define DEBUG

using std::string;
using std::cout;
using std::endl;
using std::vector;

template<typename vt, typename it>
__forceinline__ __host__ __device__        
void kernel_direct(uint idx, vt* in, vt* out, it* indices){
    it indirect_idx = indices[idx];
    if(indirect_idx < -1) {return;} // ensure read in indirection 
    out[idx] = in[idx];
}

template<typename vt, typename it>
__forceinline__ __host__ __device__        
void kernel_indirect(uint idx, vt* in, vt* out, it* indices){
    it indirect_idx = indices[idx];
    if(indirect_idx < -1) {return;}
    out[idx] = in[indirect_idx];
}

template<typename vt, typename it, bool is_indirect>
__global__        
void kernel_for_regs(uint idx, vt* in, vt* out, it* indices){
    if constexpr(is_indirect) {
        kernel_indirect<vt, it>(idx, in, out, indices);
    } else {
        kernel_direct<vt, it>(idx, in, out, indices);
    }
}

template<typename vt, typename it, bool is_indirect>
struct SimpleIndirectionKernel : public KernelCPUContext<vt, it> {
    public:
        typedef KernelCPUContext<vt, it> super;
        // name = "Array_Copy";
        int N = super::N;
        int Gsz = super::Gsz;
        int Bsz = super::Bsz;

        // Setup inputs/outputs, both data and indicies
        // Can be still in vector form, but is easier to identify explicitly with names   
        // Example
        vector<vt> & in = super::host_data[0];
        vector<vt> & out = super::host_data[1];
        vector<it> & indices = super::host_indices[0];
        vt* & d_in = super::device_data_ptrs[0];
        vt* & d_out = super::device_data_ptrs[1];
        it* & d_indices = super::device_indices_ptrs[0];
        

        int reads_per_element = 1; // Actual number 
        int writes_per_element = 1; // Actual number
        int indirect_reads_per_element = 1; // Actual number
        int total_reads;
        int total_writes;
        int total_indirect_reads;

        struct gpu_ctx {
            vt * gpu_in; 
            vt * gpu_out;
            it * gpu_indices;

            __device__        
            void operator() (uint idx){
                if constexpr(is_indirect) {
                    kernel_indirect<vt, it>(idx, gpu_in, gpu_out, gpu_indices);
                } else {
                    kernel_direct<vt, it>(idx, gpu_in, gpu_out, gpu_indices);
                }
            }
        } ctx ;

        SimpleIndirectionKernel(int n, int bs, device_context dev_ctx) 
            : super(1, 1, 1, n, bs, dev_ctx) {
            if(is_indirect){
                this->name = "SimpleIndirectionTest_Indirect";
            } else {
                this->name = "SimpleIndirectionTest_Direct";
            }
            total_reads = N * reads_per_element;
            total_writes = N * writes_per_element;
            total_indirect_reads = N * indirect_reads_per_element;
        }
        ~SimpleIndirectionKernel(){}

        void init_inputs(bool& pass) override {
            for(int i=0; i<N; ++i){
                in.push_back(i);
                out.push_back(0);
            }
        }

        void init_indices(bool& pass) override {
            for(int i=0; i<N; ++i){
                indices.push_back(i);
            }
        }

        void set_dev_ptrs() override {
            ctx.gpu_in = d_in;
            ctx.gpu_out = d_out;
            ctx.gpu_indices = d_indices;
        }

        bool local_check_result() override {
            for(int i=0; i<N; ++i){
                if(in[i] != out[i]){
                    cout << "Validation Failed at " << i << ": in="<<in[i] << " out="<< out[i] << endl;
                    return false;
                }
            }
            return true;
        }

        // No change
        void local_execute() override {
            compute_kernel<gpu_ctx><<<Gsz, Bsz>>>(N, ctx);
        }

        // No change
        void local_compute_register_usage(bool& pass) override {   
            // Kernel Registers 
            struct hipFuncAttributes funcAttrib;
            cudaErrChk(hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(*kernel_for_regs<vt),it, is_indirect>), "getting function attributes (for # registers)", pass);
            if(!pass) return;
            this->register_usage = funcAttrib.numRegs;
#ifdef DEBUG
            cout << this->name << " numRegs=" << this->register_usage << endl;
#endif
        }

};
