#include "hip/hip_runtime.h"
#pragma once 
/**
 * @file kernel_context.cu
 * @author Dalton Winans-Pruitt (daltonrpruitt@gmail.com)
 * @brief Provides context information for GPU kernel execution of driver
 * @version 0.1
 * @date 2022-01-27
 * 
 */

#include <vector>

#include <hip/hip_runtime.h>
#include <local_cuda_utils.h>

#define DEBUG

using std::string;
using std::to_string;
using std::cout;
using std::endl;
using std::vector;


template<typename kernel_ctx_t>
__global__
void compute_kernel(int N, kernel_ctx_t ctx) {
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= N) return;
    ctx(idx);
}



template<typename vt, typename it>
struct KernelCPUContext {
    public:
        string name;
        int N=-1;
        int Bsz=-1;
        int Gsz=-1;
        int num_in_data=-1;
        int num_out_data=-1;
        int num_total_data=-1;
        int num_indices=-1;

        vector<vector<vt>> host_data{(unsigned long)num_total_data};
        vector<vt *> device_data_ptrs{(unsigned long)num_total_data};

        bool okay = true;
        
        vector<vector<it>> host_indices{(unsigned long)num_indices};
        vector<it *> device_indices_ptrs{(unsigned long)num_indices};


        void free(){
            for(vt* ptr : device_data_ptrs) hipFree(ptr);
            for(it* ptr : device_indices_ptrs) hipFree(ptr);
        }

        virtual void init_inputs() {};
        virtual void init_indices() {};

        KernelCPUContext(int in, int out, int indices, int n, int bs)
            : num_in_data(in), num_out_data(out), num_indices(indices), 
            num_total_data(in+out), N(n), Bsz(bs), Gsz( (n+bs-1)/bs )  {}

        void init(){
            
            init_inputs();
            init_indices();

            device_data_ptrs.resize(num_total_data);

            bool pass = true;
            for(int i=0; i < num_total_data; ++i) {
                cudaErrChk(hipMalloc((void **)&device_data_ptrs[i], N * sizeof(vt)),"device_data_ptrs["+to_string(i)+"] mem allocation", pass);
                if(!pass) break;
            }
            
            if(pass) {
               for(int i=0; i < num_in_data; ++i) {
                    cudaErrChk(hipMemcpy(device_data_ptrs[i], host_data[i].data(), N * sizeof(vt), hipMemcpyHostToDevice), "copy host_data["+to_string(i)+"] to device_data_ptrs["+to_string(i)+"]", pass);                
                    if(!pass) break;
                }
            }

            if(!pass) {free(); okay = false;}
            else { set_dev_ptrs(); }
        }

        ~KernelCPUContext(){
            free();            
        }

        virtual void set_dev_ptrs() {}

        virtual void local_execute() {}

        void execute() {
            if(!okay) return;
            local_execute();
            
            bool pass = true;
            for(int i=num_in_data; i < num_total_data; ++i) {
                cudaErrChk(hipMemcpy(host_data[i].data(), device_data_ptrs[i], N * sizeof(vt), hipMemcpyDeviceToHost),"copying device_data_ptrs["+to_string(i)+"] to host_data["+to_string(i)+"]", pass);
            }
            
            if(!pass) {free(); okay = false;}
        }

        virtual bool local_check_result() = 0;

        bool check_result() {
            if(!okay){
                cout << "Cannot check "<< name << " due to previous failure!" << endl;
                return false;
            };
            return local_check_result();
        }

        void run() {
            init();
            execute();
        }
        
        bool run_and_check() {
            run();
            return check_result();     
        }

};
