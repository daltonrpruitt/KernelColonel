#include "hip/hip_runtime.h"
#pragma once 
/**
 * @file kernel_context.cu
 * @author Dalton Winans-Pruitt (daltonrpruitt@gmail.com)
 * @brief Provides context information for GPU kernel execution of driver
 * @version 0.1
 * @date 2022-01-27
 * 
 */

#include <local_cuda_utils.h>
#include <device_props.h>

#include <vector>
#include <algorithm>

#include <hip/hip_runtime.h>

using std::string;
using std::to_string;
using std::cout;
using std::endl;
using std::vector;


template<typename kernel_ctx_t>
__global__
void compute_kernel(unsigned long long N, kernel_ctx_t ctx) {
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= N) return;
    ctx(idx);
}

template<typename gpu_ctx>
inline
float local_execute_template(int N, int Gsz, int Bsz, int shdmem_usage, device_context* dev_ctx, gpu_ctx ctx) {
    if(dev_ctx->props_.major >= 7) {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(compute_kernel<gpu_ctx>));
        int shmem = dev_ctx->props_.sharedMemPerMultiprocessor-1024-attr.sharedSizeBytes;
        hipFuncSetAttribute(reinterpret_cast<const void*>(compute_kernel<gpu_ctx>), hipFuncAttributeMaxDynamicSharedMemorySize, shmem);
        cudaPrintLastError();
    }
    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);

    hipEventRecord(start);
    compute_kernel<gpu_ctx><<<Gsz, Bsz, shdmem_usage>>>(N, ctx);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    cudaPrintLastError();

    float time = 0;
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start); hipEventDestroy(stop);
    return time; 
}


template<typename vt, typename it>
struct KernelCPUContext {
    public:
        string name;
        unsigned long long N=0;
        int Bsz=-1;
        int Gsz=-1;
        int num_in_data=-1;
        int num_out_data=-1;
        int num_total_data=-1;
        int num_indices=-1;
        
        bool okay = true;
        bool initialized = true;

        size_t shared_memory_usage=0;
        int register_usage=-1;
        int max_blocks_simultaneous_per_sm=-1;

        int total_data_reads;
        int total_index_reads;
        int total_writes;

        device_context* dev_ctx;

        vector<vector<vt>> host_data{(unsigned long)num_total_data};
        vector<vt *> device_data_ptrs{(unsigned long)num_total_data};

        
        vector<vector<it>> host_indices{(unsigned long)num_indices};
        vector<it *> device_indices_ptrs{(unsigned long)num_indices};


        void free(){
            for(vt* ptr : device_data_ptrs)     { hipFree(ptr); ptr = nullptr; }
            for(it* ptr : device_indices_ptrs)  { hipFree(ptr); ptr = nullptr; }
        }
        
        void uninit() {
            if(!initialized) {return;}
            free();
            for(int i=0; i<num_total_data; ++i) { host_data[i].clear(); }
            for(int i=0; i<num_indices; ++i) { host_indices[i].clear(); }
            }

        virtual void init_inputs(bool& pass) {};
        virtual void init_indices(bool& pass) {};

        KernelCPUContext(int in, int out, int indices, unsigned long long n, int bs, device_context* d_ctx, int shd_mem_alloc=0)
            : num_in_data(in), num_out_data(out), num_indices(indices), 
            num_total_data(in+out), N(n), Bsz(bs), Gsz( (n+bs-1)/bs ), dev_ctx(d_ctx), shared_memory_usage(shd_mem_alloc) {
            }

        bool init(){
            bool pass = true;

            compute_max_simultaneous_blocks(pass);
            if(pass) init_inputs(pass);
            if(pass) init_indices(pass);

            if(pass){

                device_data_ptrs.resize(num_total_data);

                for(int i=0; i < num_total_data; ++i) {
                    cudaErrChk(hipMalloc((void **)&device_data_ptrs[i], N * sizeof(vt)),"device_data_ptrs["+to_string(i)+"] mem allocation", pass);
                    if(!pass) break;
                }
                
                if(pass) {
                    for(int i=0; i < num_in_data; ++i) {
                        cudaErrChk(hipMemcpy(device_data_ptrs[i], host_data[i].data(), N * sizeof(vt), hipMemcpyHostToDevice), "copy host_data["+to_string(i)+"] to device_data_ptrs["+to_string(i)+"]", pass);                
                        if(!pass) break;
                    }
                }

                for(int i=0; i < num_indices; ++i) {
                    cudaErrChk(hipMalloc((void **)&device_indices_ptrs[i], N * sizeof(it)),"device_indices_ptrs["+to_string(i)+"] mem allocation", pass);
                    if(!pass) break;
                }

                if(pass) {
                    for(int i=0; i < num_indices; ++i) {
                        cudaErrChk(hipMemcpy(device_indices_ptrs[i], host_indices[i].data(), N * sizeof(it), hipMemcpyHostToDevice), "copy host_indices["+to_string(i)+"] to device_indices_ptrs["+to_string(i)+"]", pass);                
                        if(!pass) break;
                    }
                }

                if(pass) { set_dev_ptrs(); }
            }

            if(!pass) {
                free(); 
                okay = false;
                cerr<<"Error in initializing "<<this->name << "for N="<<this->N<<" Bsz="<<this->Bsz<<" !" << endl;
            }
            return pass;
        }

        ~KernelCPUContext(){
            free();            
        }

        virtual void output_config_info() {
            cout << name << endl; 
        }

        virtual void set_dev_ptrs() {}

        virtual float local_execute() = 0;

        float execute() {
            if(!okay) return -1.0;

            float time = local_execute();

            bool pass = true;
            for(int i=num_in_data; i < num_total_data; ++i) {
                cudaErrChk(hipMemcpy(host_data[i].data(), device_data_ptrs[i], N * sizeof(vt), hipMemcpyDeviceToHost),"copying device_data_ptrs["+to_string(i)+"] to host_data["+to_string(i)+"]", pass);
            }
            
            if(!pass) {free(); okay = false; time = -1.0;}
            return time;
        }

        virtual bool local_check_result() = 0;

        bool check_result() {
            if(!okay){
                cout << "Cannot check "<< name << " due to previous failure!" << endl;
                return false;
            };
            return local_check_result();
        }

        float run() {
            if(!initialized) {
                if(!init()) return -1.0;
            }
            return execute();
        }
        
        bool run_and_check() {
            run(); // ignore time
            return check_result();     
        }

    virtual void local_compute_register_usage(bool& pass) = 0;

    void compute_max_simultaneous_blocks(bool& pass) {
        local_compute_register_usage(pass);
        if(!pass) { okay = false; return;}
        int due_to_block_size = (int) floor(dev_ctx->props_.maxThreadsPerMultiProcessor / Bsz); 
        int due_to_registers =  (int) floor(dev_ctx->props_.regsPerMultiprocessor / (register_usage * Bsz));
        max_blocks_simultaneous_per_sm = std::min({due_to_block_size, 
                                            due_to_registers, dev_ctx->props_.maxBlocksPerMultiProcessor});

    }

    vector<int> shared_memory_allocations() {
        vector<int> alloc_amounts; 
        bool pass = true;
        if(max_blocks_simultaneous_per_sm < 0) compute_max_simultaneous_blocks(pass);
        if(!pass) { 
            okay = false;  
            alloc_amounts.push_back(-1);
            return alloc_amounts;
        }
        int max_shd_mem_per_block = dev_ctx->props_.sharedMemPerBlock;
        int max_shd_mem_per_proc = dev_ctx->props_.sharedMemPerMultiprocessor;

        int min_blocks_due_to_shd_mem = max_shd_mem_per_proc / max_shd_mem_per_block;

        for(int i=min_blocks_due_to_shd_mem; i < max_blocks_simultaneous_per_sm ; i+=1) {
            int sm_alloc = std::min((max_shd_mem_per_proc / i - 256) / 256 * 256, max_shd_mem_per_block);
            alloc_amounts.push_back(sm_alloc);
        }
        return alloc_amounts;
    }

    float get_occupancy() {
        int max_blocks_shared_mem;
        if(shared_memory_usage == 0) {
            max_blocks_shared_mem = dev_ctx->props_.maxBlocksPerMultiProcessor;
        } else {
            max_blocks_shared_mem = dev_ctx->props_.sharedMemPerMultiprocessor / shared_memory_usage;
        }
        int max_blocks_simul = std::min(max_blocks_simultaneous_per_sm, max_blocks_shared_mem);
        int num_threads_simul = max_blocks_simul * Bsz; 
        return float(num_threads_simul) / float(dev_ctx->props_.maxThreadsPerMultiProcessor);
    }

    void print_register_usage() {
        bool pass = true; 
        if(register_usage < 0) { 
            local_compute_register_usage(pass);
        }
        if(!pass) {cerr << "Cannot get register usage for " << name << "!" << endl;}
        else { cout << name << " register usage = " << register_usage << endl;}
    }

    unsigned long long get_total_bytes_processed() {
        return ( total_data_reads+ total_writes)*sizeof(vt) +  total_index_reads*sizeof(it);
    }

};
