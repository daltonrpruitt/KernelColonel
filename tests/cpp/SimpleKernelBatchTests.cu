#include "hip/hip_runtime.h"
/**
 * @file SimpleKernelRunSetTests.cu
 * @author Dalton Winans-Pruitt (daltonrpruitt@gmail.com)
 * @brief Set of unit/integration tests for the IKernelExecution using a kernel from a file
 *
 */

#include <gtest/gtest.h>
#include <gmock/gmock.h>

#include <iostream>
#include <string>
#include <memory>

// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime_api.h>


#include "data/SimpleKernelData.hpp"
#include "execution/SimpleKernelExecution.hpp"
#include "execution/SimpleKernelRunSet.hpp"
#include "execution/SimpleKernelBatch.hpp"
#include "execution/JitifyCache.cuh"
#include <execution/kc_jitify.hpp>

#include "utils/utils.hpp"

namespace kc = KernelColonel;

template<typename vt = double>
std::vector<vt> whole_numbers(unsigned int length)
{
    std::vector<vt> out;
    for(unsigned int i=0; i < length; i++)
        out.push_back(static_cast<vt>(i));
    return out;
}


static std::string copy_kernel_program_name  { "simple_copy_kernel" };
static const std::string simple_copy_kernel_source_string =
    copy_kernel_program_name + 
    std::string(R"(
    template<typename value_t, typename index_t> 
    struct SimpleKernelData_gpu_data_s
    {
        value_t* input = nullptr;
        value_t* output = nullptr;
        index_t* indices = nullptr;
    };

    template<typename vt, typename it>
    __global__
    void simple_copy_kernel(unsigned int N, SimpleKernelData_gpu_data_s<vt,it> gpu_data) {
        // unsigned int x = blockIdx.x * blockSize.x + threadIdx.x;
        if(blockIdx.x != 0 || threadIdx.x != 0) return;
        for( int i=0; i<N; ++i ) {
            gpu_data.output[i] = gpu_data.input[i];
            // if(i<10) printf("At i=%d input[i]=%f, output[i]=%f\n",i, gpu_data.input[i], gpu_data.output[i]);
        }
    })");

class SimpleKernelBatchTests : public ::testing::Test // : public IKernelExecution<...>
{
public:
    SimpleKernelBatchTests() = default;
    ~SimpleKernelBatchTests() = default;

    void SetUp()
    {

        data_ptr = std::make_shared<kc::SimpleKernelData<>>(data_size);
        data_ptr->setInitInputsFunc(whole_numbers<>);
        program = kc::globalJitifyCache().program(simple_copy_kernel_source_string);
    }

    void TearDown()
    {
        data_ptr.reset();
    }

protected:
    unsigned long long data_size = 5;
    std::shared_ptr<kc::SimpleKernelData<>> data_ptr;
    jitify::Program program; 
};

TEST_F(SimpleKernelBatchTests, CreateWithData)
{
    kc::SimpleKernelBatch<> batch(data_ptr);
}



TEST_F(SimpleKernelBatchTests, AddRunSets)
{
    auto check_lambda = [&](const auto &input, const auto &output, const auto &indices)
    {
        if (input.size() != output.size())
            return false;
        for (int i = 0; i < input.size(); i++)
            if (input[i] != output[i])
            {
                std::cout << "data differs at i="<<i<<"with in[i]="<<input[i] << " and out[i]=" <<output[i] << std::endl;
                return false;
            }

        return true;
    };


    kc::SimpleKernelBatch<> batch(data_ptr);
    auto exec_ptr = std::make_shared<kc::SimpleKernelExecution<>>("simple_copy_kernel", program, check_lambda);
    
    std::vector<std::shared_ptr<kc::SimpleKernelRunSet<>>> run_set_ptrs;

    for(int i=1; i<=8; i*=2)
    {
        for(int j=1; j<=8; j*=2)
        {
            auto run_set_ptr = std::make_shared<kc::SimpleKernelRunSet<>>(exec_ptr, data_ptr, dim3(i), dim3(j));
            ASSERT_TRUE(batch.add_run_set(run_set_ptr));
        }
    }
}
/*
    kc::SimpleKernelRunSet<> run_set(exec_ptr, data_ptr, grid, block, times_run);


    auto check_lambda = [&](const auto &input, const auto &output, const auto &indices)
    {
        if (input.size() != output.size())
            return false;
        for (int i = 0; i < input.size(); i++)
            if (input[i] != output[i])
            {
                std::cout << "data differs at i="<<i<<"with in[i]="<<input[i] << " and out[i]=" <<output[i] << std::endl;
                return false;
            }

        return true;
    };


    dim3 grid(1);
    dim3 block(1);

    unsigned int times_run = 5;

    ASSERT_TRUE(run_set.check_and_run_all());
    auto times = run_set.get_run_timings();
    ASSERT_EQ(times.size(), times_run);

    std::cout << "Executing " << "simple_copy_kernel" << " with grid=" << grid << " and block=" << block << " for " << times_run << " times took the following ms timings: " << times << std::endl;
}
*/