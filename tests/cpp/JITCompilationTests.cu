#include "hip/hip_runtime.h"
/**
 * @file JITCompilationTests.cu
 * @author Dalton Winans-Pruitt (daltonrpruitt@gmail.com)
 * @brief Set of unit/integration tests for use of JIT compilation of kernels
 * 
 * Meant to help alleviate need for compile-time knowledge of various kernel parameters, such as those 
 * discussed in [this Medium article](https://medium.com/gpgpu/cuda-jit-compilation-1fb4950c67bb) and in 
 * [this GTC 2017 talk](https://on-demand.gputechconf.com/gtc/2017/videos/s7716-barsdell-ben-jitify.mp4).
 * Uses the repo discussed in the talk: [jitify](https://github.com/NVIDIA/jitify). 
 * The jitify repo has a BSD-3-Clause license. 
 * 
 * A lot of this is taken straight from the example in jitify (see 
 * [here](https://github.com/NVIDIA/jitify/blob/master/jitify_example.cpp)).
 * 
 * @version 0.1
 * @date 2022-12-03
 * 
 * @copyright Copyright (c) 2022
 * 
 */

#include <gtest/gtest.h>
#include <gmock/gmock.h>

#include <iostream>
#include <string>
#include <tuple>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#ifdef LINUX  // Only supported by gcc on Linux (defined in Makefile)
#define JITIFY_ENABLE_EMBEDDED_FILES 1
#endif

#define VERBOSE
#ifdef VERBOSE
#define JITIFY_PRINT_INSTANTIATION 1
#define JITIFY_PRINT_SOURCE 1
#define JITIFY_PRINT_LOG 1
#define JITIFY_PRINT_PTX 1
#define JITIFY_PRINT_LINKER_LOG 1
#define JITIFY_PRINT_LAUNCH 1
#endif // VERBOSE
#include "jitify.hpp"

#define CHECK_CUDA(call)                                                  \
  do {                                                                    \
    if (call != hipSuccess) {                                           \
      const char* str;                                                    \
      hipDrvGetErrorName(call, &str);                                         \
      std::cout << "(CUDA) returned " << str;                             \
      std::cout << " (" << __FILE__ << ":" << __LINE__ << ":" << __func__ \
                << "())" << std::endl;                                    \
      FAIL() << "Experienced above CUDA error!";                          \
    }                                                                     \
  } while (0)


template <typename T>
bool are_close(T in, T out) {
  return fabs(in - out) <= 1e-5f * fabs(in);
}


TEST(JITCompilationTest, SimpleProgram) {
    const char* program_source =
        "my_program\n"
        "template<int N, typename T>\n"
        "__global__\n"
        "void my_kernel(T* data) {\n"
        "    T data0 = data[0];\n"
        "    for( int i=0; i<N-1; ++i ) {\n"
        "        data[0] *= data0;\n"
        "    }\n"
        "}\n";
    static jitify::JitCache kernel_cache;
    jitify::Program program = kernel_cache.program(program_source, 0);
    
    using T = float;

    T h_data = 5;
    T* d_data;
    hipMalloc((void**)&d_data, sizeof(T));
    hipMemcpy(d_data, &h_data, sizeof(T), hipMemcpyHostToDevice);
    dim3 grid(1);
    dim3 block(1);
    using jitify::reflection::type_of;
    CHECK_CUDA(program.kernel("my_kernel")
                    .instantiate(3, type_of(*d_data))
                    .configure(grid, block)
                    .launch(d_data));
    hipMemcpy(&h_data, d_data, sizeof(T), hipMemcpyDeviceToHost);
    hipFree(d_data);
    std::cout << h_data << std::endl;
    ASSERT_TRUE(are_close(h_data, 125.f));
}

namespace std
{
    template<typename T>
    std::ostream& operator<<(std::ostream& os, const std::vector<T>& vec){
        os << "< " << std::to_string(vec[0]);
            for (int i=1; i<vec.size() && i < 10; ++i) { os << ", " << vec[i]; }
        if(vec.size() > 10) { os << " ... "; }
        os << ">" << std::endl;
        return os;
    }
} // namespace std

/**
 * @brief What I need to test for here.
 * 
 * - can pass template parameter pack through to the kernel instantiation
 * 
 */

template<int N, typename... Ts> using NthTypeOf =
        typename std::tuple_element<N, std::tuple<Ts...>>::type;

static jitify::JitCache kernel_cache;

template<typename ...io_types>
class simple_kernel
{
    const char* program_source =
        "my_program\n"
        "template<typename in_t, typename out_t>\n"
        "__global__\n"
        "void single_thread_copy(unsigned int N, in_t* in, out_t* out) {\n"
        "    for( int i=0; i<N; ++i ) {\n"
        "        out[i] = in[i];\n" // should auto cast? 
        "    }\n"
        "}\n";
    bool m_compiled = false;
    jitify::Program m_program;

  public:
    void compile() {
        m_program = kernel_cache.program(program_source, 0);
        m_compiled = true;
    }

    std::vector<NthTypeOf<1,io_types...>> run(int N) {
        // unsigned int N = 5;
        if (!m_compiled) { compile(); }
        using in_t = NthTypeOf<0,io_types...>;
        using out_t = NthTypeOf<1,io_types...>;


        std::vector<in_t> h_input; for(int i=0; i<N; ++i) { h_input.push_back(i); }
        std::vector<out_t> h_output(N, 0);
        in_t* d_input;
        out_t* d_output;
        std::tuple<io_types...> tpl;

        hipMalloc((void**)&d_input, sizeof(in_t)*N);
        hipMalloc((void**)&d_output, sizeof(out_t)*N);

        hipMemcpy(d_input, &h_input[0], sizeof(in_t)*N, hipMemcpyHostToDevice);
        dim3 grid(1);
        dim3 block(1);
        using jitify::reflection::reflect_template;
        using jitify::reflection::reflect;
        std::cout<< reflect_template<io_types...>() << std::endl;

        auto instance = m_program.kernel("single_thread_copy").instantiate<io_types...>();
        auto launcher = instance.configure(grid, block);
        launcher.safe_launch(N, d_input, d_output);

        hipMemcpy(&h_output[0], d_output, sizeof(out_t)*N, hipMemcpyDeviceToHost);
        std::cout << h_output;
        hipFree(d_input);
        hipFree(d_output);
        return h_output;

    }
};

TEST(JITCompilationTest, ParameterPackPassToKernel) {
    
    // using in_t = int;
    // using out_t = int;

    unsigned int N = 5;
    // std::vector<in_t> h_input; for(int i=0; i<N; ++i) { h_input.push_back(i); }
    // std::vector<out_t> h_output(N, 0);
    // in_t* d_input;
    // out_t* d_output;

    // hipMalloc((void**)&d_input, sizeof(in_t));
    // hipMalloc((void**)&d_output, sizeof(out_t));

    // hipMemcpy(d_input, &h_input, sizeof(in_t)*N, hipMemcpyHostToDevice);
    // dim3 grid(1);
    // dim3 block(1);
    // using jitify::reflection::type_of;
    // CHECK_CUDA(program.kernel("single_thread_copy")
    //                 .instantiate(type_of(*d_input), type_of(*d_output))
    //                 .configure(grid, block)
    //                 .launch(N, d_input, d_output));
    // hipMemcpy(&h_output, d_output, sizeof(out_t)*N, hipMemcpyDeviceToHost);
    // hipFree(d_input);
    // hipFree(d_output);
    simple_kernel<double, int> first_kernel;
    auto h_output = first_kernel.run(N);
    std::cout << h_output;

    for (int i=0; i<N; ++i) {
        ASSERT_TRUE(std::abs(h_output[i] - i) < 1e-5);
    }
}
